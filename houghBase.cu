#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

// Memoria constante respecto a los senos y cosenos
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];


//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

// Memoria constante
__global__ void GPU_HoughTranConst(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  //TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

}

// Memoria compartida
__global__ void GPU_HoughTranShared(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  //TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //usar memoria compartida para el acumulador
  int locID = threadIdx.x;
  __shared__ int localH[degreeBins * rBins]; 
  int i;

  for(i = locID; i < degreeBins * rBins; i += blockDim.x)
    localH[i] = 0;
  

  __syncthreads();

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

  __syncthreads();

  for(i = locID; i < degreeBins * rBins; i += blockDim.x){
    atomicAdd(acc + i, localH[i]);
  }

}

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  //TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

  //TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
  //utilizar operaciones atomicas para seguridad
  //faltara sincronizar los hilos del bloque en algunos lados

}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  // Los timers para medicion
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //float* d_Cos;
  //float* d_Sin;

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // TODO eventualmente volver memoria global
  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);

  //CUDA medicion de tiempoS
  hipEventRecord(start);
  //GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);
  GPU_HoughTranShared <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);
  hipEventRecord(stop);
  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float program_timer = 0;
  hipEventElapsedTime(&program_timer, start, stop);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");
  printf(" Time elapsed during the Hough Formula: %f\n\n", program_timer);

  // cleanup de la memorisa usada

  free (pcCos);
  free (pcSin);
  free (h_hough);
  hipFree ((void *) d_in);
  hipFree ((void *) d_hough);
  hipDeviceReset();
  



  return 0;
}

